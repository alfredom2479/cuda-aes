#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "string.h"

#define DEFAULT_FILENAME "gary.ppm"

#define BLOCK_SIZE 256

unsigned char *read_ppm(char *filename, int *xsize, int *ysize, int *maxval){
    FILE *fp = fopen(filename, "rb");
    if(!fp){
        fprintf(stderr, "Error: '%s' cannot be opened\n", filename);
        return NULL;
    }
    char first_line[4];
    fgets(first_line,4,fp);
    if(strcmp(first_line, "P6\n")){
        fprintf(stderr, "Error: '%s' not in P6 format\n", filename);
        return NULL;
    }
    char temp;
    while((temp = fgetc(fp)) == '#'){
        fscanf(fp, "%*[^\n]\n");
    }
    ungetc(temp,fp);
    fscanf(fp, "%d %d\n%d\n", xsize, ysize, maxval);
    
    unsigned int *pic  = (unsigned int*)malloc((*xsize)*(*ysize)*sizeof(unsigned int));
    unsigned char *ppic = (unsigned char*)malloc((*xsize)*(*ysize)*sizeof(unsigned char)*3);
    for(int i = 0; i < (*xsize)*(*ysize); i++){
        unsigned char buf[3];
        fread((void*)&buf[0], 3, 1, fp);
        //printf("%x,%x,%x\n",buf[0],buf[1],buf[2]);
        //pic[i] = buf[0];
        ppic[i*3]= buf[0];
        ppic[i*3+1] = buf[1];
        ppic[i*3+2] = buf[2];
        
    }
    return ppic;
}

void write_ppm(unsigned char *filename, int xsize, int ysize, int maxval, unsigned char *pic){
    FILE *fp;
    
    fp = fopen((char *)filename, "wb");
    if(!fp){
        fprintf(stderr, "FAILED TO OPEN FILE '%s' for writing\n", filename);
        exit(-1);
    }
    
    fprintf(fp, "P6\n");
    fprintf(fp, "%d %d\n%d\n", xsize, ysize, maxval);
    
    int numpix = xsize * ysize;
    for(int i=0; i < numpix; i++){
        //unsigned char uc = (unsigned char) pic[i];
        fprintf(fp, "%c%c%c", pic[i*3], pic[i*3+1], pic[i*3+2]);
    }
    
    fclose(fp);
}


//__global__ void aes(unsigned int* pic, int* r_pic, int xsize, int ysize, int thresh){
__global__ void aes_naive(unsigned char* pic, unsigned char* r_pic, unsigned char* keys, unsigned char* sbox){
    //current plan is to prepend a 0x00 per pixel so that we can have split up data evenly
    //4 pixels per thread
    //first 16 bytes will contain the number of extra bytes that have been appended to the array
    unsigned char res[4];
    unsigned char mcMatrix[] ={
        0x02, 0x03, 0x01, 0x01,
        0x01, 0x02, 0x03, 0x01,
        0x01, 0x01, 0x02, 0x03,
        0x03, 0x01, 0x01, 0x02
    };
    
    int tx = (blockIdx.x*BLOCK_SIZE+threadIdx.x)*16;
    unsigned char temp, temp0, temp1;
    
    //make local copy to
    unsigned char l_pic[16];
    
    for(int i = 0; i< 16; i++){
        l_pic[i] = pic[tx+i];    
    }
    //initial add round key
    for(int i = 0; i < 16; i++){
        l_pic[i] = l_pic[i] ^ keys[i];
    }
    //The main aes loop
    for(int i = 0; i < 10; i++){
        //sub bytes
        for(int j = 0; j < 16; j++){
            l_pic[j] = sbox[l_pic[j]];
        }
        //shift rows
        temp = l_pic[1];
        l_pic[1] = l_pic[1+4];
        l_pic[1+4] = l_pic[1+8];
        l_pic[1+8] = l_pic[1+12];
        l_pic[1+12] = temp;
        
        temp = l_pic[2];
        l_pic[2] = l_pic[2+8];
        l_pic[2+8] = temp;
        temp = l_pic[2+4];
        l_pic[2+4] = l_pic[2+12];
        l_pic[2+12] = temp;
        
        temp = l_pic[3];
        l_pic[3] = l_pic[3+12];
        l_pic[3+12] = l_pic[3+8];
        l_pic[3+8] = l_pic[3+4];
        l_pic[3+4] = temp;
        
        //mix columns
        if(i<9){
            for(int j=0; j < 4; j++){
                res[0] = 0x00;
                res[1] = 0x00;
                res[2] = 0x00;
                res[3] = 0x00;
                
                for(int k = 0; k < 4; k++){
                    //res[0] gf2_mul
                    temp = 0x00; 
                    temp0 = mcMatrix[0+k];
                    temp1 = l_pic[j*4+k];
                    for(;temp1;temp1>>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[0] = res[0] ^ temp;
                    
                    //res[1] gf2_mul
                    temp = 0x00; 
                    temp0 = mcMatrix[4+k];
                    temp1 = l_pic[j*4+k];
                    for(;temp1;temp1 >>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[1] = res[1] ^ temp;
                    
                    //res[2] gf2_mul
                    temp = 0x00; 
                    temp0 = mcMatrix[8+k];
                    temp1 = l_pic[j*4+k];
                    for(;temp1;temp1 >>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[2] = res[2] ^ temp;
                    
                    //res[3] gf2_mul
                    temp = 0x00; 
                    temp0 = mcMatrix[12+k];
                    temp1 = l_pic[j*4+k];
                    for(;temp1;temp1 >>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[3] = res[3] ^ temp;
                }
                l_pic[(j*4)+0] = res[0];
                l_pic[(j*4)+1] = res[1];
                l_pic[(j*4)+2] = res[2];
                l_pic[(j*4)+3] = res[3];
            }
            
        }
        //add round key
        for(int j = 0; j < 16; j++){
            l_pic[j] = l_pic[j] ^ keys[((i+1)*16)+j];
        }
       //end of of current round of encryption
    }
    //end of all encryption
    //move local bytes back to global memory
    for(int i = 0; i < 16; i++){
        r_pic[tx+i] =  l_pic[i];
    }
}

__global__ void aes(unsigned char* pic, unsigned char* r_pic, unsigned char* keys, unsigned char* sbox){
    //current plan is to prepend a 0x00 per pixel so that we can have split up data evenly
    //4 pixels per thread
    //first 16 bytes will contain the number of extra bytes that have been appended to the array
    
    __shared__ unsigned char keys_s[176];
    __shared__ unsigned char sbox_s[256];
    
    sbox_s[threadIdx.x] = sbox_s[threadIdx.x];
    
    if(threadIdx.x < 176){
        keys_s[threadIdx.x] = keys_s[threadIdx.x];
    }
    unsigned char res[4];
    unsigned char mcMatrix[] ={
        0x02, 0x03, 0x01, 0x01,
        0x01, 0x02, 0x03, 0x01,
        0x01, 0x01, 0x02, 0x03,
        0x03, 0x01, 0x01, 0x02
    };
    
    int tx = (blockIdx.x*BLOCK_SIZE+threadIdx.x)*16;
    unsigned char temp, temp0, temp1;
    
    //make local copy to
    unsigned char l_pic[16];
    
    for(int i = 0; i< 16; i++){
        l_pic[i] = pic[tx+i];    
    }
    //initial add round key
    for(int i = 0; i < 16; i++){
        l_pic[i] = l_pic[i] ^ keys[i];
    }
    //The main aes loop
    for(int i = 0; i < 10; i++){
        //sub bytes
        for(int j = 0; j < 16; j++){
            l_pic[j] = sbox_s[l_pic[j]];
        }
        //shift rows
        temp = l_pic[1];
        l_pic[1] = l_pic[1+4];
        l_pic[1+4] = l_pic[1+8];
        l_pic[1+8] = l_pic[1+12];
        l_pic[1+12] = temp;
        
        temp = l_pic[2];
        l_pic[2] = l_pic[2+8];
        l_pic[2+8] = temp;
        temp = l_pic[2+4];
        l_pic[2+4] = l_pic[2+12];
        l_pic[2+12] = temp;
        
        temp = l_pic[3];
        l_pic[3] = l_pic[3+12];
        l_pic[3+12] = l_pic[3+8];
        l_pic[3+8] = l_pic[3+4];
        l_pic[3+4] = temp;
        
        //mix columns
        if(i<9){
            for(int j=0; j < 4; j++){
                res[0] = 0x00;
                res[1] = 0x00;
                res[2] = 0x00;
                res[3] = 0x00;
                
                for(int k = 0; k < 4; k++){
                    //res[0] gf2_mul
                    temp = 0x00; 
                    temp0 = mcMatrix[0+k];
                    temp1 = l_pic[j*4+k];
                    for(;temp1;temp1>>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[0] = res[0] ^ temp;
                    
                    //res[1] gf2_mul
                    temp = 0x00; 
                    temp0 = mcMatrix[4+k];
                    temp1 = l_pic[j*4+k];
                    for(;temp1;temp1 >>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[1] = res[1] ^ temp;
                    
                    //res[2] gf2_mul
                    temp = 0x00; 
                    temp0 = mcMatrix[8+k];
                    temp1 = l_pic[j*4+k];
                    for(;temp1;temp1 >>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[2] = res[2] ^ temp;
                    
                    //res[3] gf2_mul
                    temp = 0x00; 
                    temp0 = mcMatrix[12+k];
                    temp1 = l_pic[j*4+k];
                    for(;temp1;temp1 >>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[3] = res[3] ^ temp;
                }
                l_pic[(j*4)+0] = res[0];
                l_pic[(j*4)+1] = res[1];
                l_pic[(j*4)+2] = res[2];
                l_pic[(j*4)+3] = res[3];
            }
            
        }
        //add round key
        for(int j = 0; j < 16; j++){
            l_pic[j] = l_pic[j] ^ keys_s[((i+1)*16)+j];
        }
       //end of of current round of encryption
    }
    //end of all encryption
    //move local bytes back to global memory
    for(int i = 0; i < 16; i++){
        r_pic[tx+i] =  l_pic[i];
    }
}


__global__ void aes_moreWork(unsigned char* pic, unsigned char* r_pic, unsigned char* keys, unsigned char* sbox, int size){
    //current plan is to prepend a 0x00 per pixel so that we can have split up data evenly
    //4 pixels per thread
    //first 16 bytes will contain the number of extra bytes that have been appended to the array
    
    __shared__ unsigned char keys_s[176];
    __shared__ unsigned char sbox_s[256];
    
    sbox_s[threadIdx.x] = sbox_s[threadIdx.x];
    
    if(threadIdx.x < 176){
        keys_s[threadIdx.x] = keys_s[threadIdx.x];
    }
    unsigned char res[8];
    unsigned char mcMatrix[] ={
        0x02, 0x03, 0x01, 0x01,
        0x01, 0x02, 0x03, 0x01,
        0x01, 0x01, 0x02, 0x03,
        0x03, 0x01, 0x01, 0x02
    };
    
    //int tx = (blockIdx.x*BLOCK_SIZE+threadIdx.x)*16;
    int tx = (blockIdx.x*BLOCK_SIZE*threadIdx.x)*32;
    
    unsigned char temp, temp0, temp1;
    
    //make local copy to
    unsigned char l_pic[32];
    
    //printf("!");
    if(tx+32 < size){
    for(int i = 0; i< 32; i++){
        l_pic[i] = pic[tx+i];    
    }
    
    //initial add round key
    for(int i = 0; i < 16; i++){
        temp = keys[i];
        //l_pic[i] = l_pic[i] ^ keys[i];
        l_pic[i] = l_pic[i] ^ temp;
        l_pic[16+i] = l_pic[16+i] ^ temp;
    }
    //The main aes loop
    for(int i = 0; i < 10; i++){
        //sub bytes
        for(int j = 0; j < 32; j++){
            l_pic[j] = sbox_s[l_pic[j]];
        }
        //shift rows
        temp = l_pic[1];
        l_pic[1] = l_pic[1+4];
        l_pic[1+4] = l_pic[1+8];
        l_pic[1+8] = l_pic[1+12];
        l_pic[1+12] = temp;
        
        temp = l_pic[2];
        l_pic[2] = l_pic[2+8];
        l_pic[2+8] = temp;
        temp = l_pic[2+4];
        l_pic[2+4] = l_pic[2+12];
        l_pic[2+12] = temp;
        
        temp = l_pic[3];
        l_pic[3] = l_pic[3+12];
        l_pic[3+12] = l_pic[3+8];
        l_pic[3+8] = l_pic[3+4];
        l_pic[3+4] = temp;
        
        //2nd matrix
        temp = l_pic[16+1];
        l_pic[16+1] = l_pic[16+1+4];
        l_pic[16+1+4] = l_pic[16+1+8];
        l_pic[16+1+8] = l_pic[16+1+12];
        l_pic[16+1+12] = temp;
        
        temp = l_pic[16+2];
        l_pic[16+2] = l_pic[16+2+8];
        l_pic[16+2+8] = temp;
        temp = l_pic[16+2+4];
        l_pic[16+2+4] = l_pic[16+2+12];
        l_pic[16+2+12] = temp;
        
        temp = l_pic[16+3];
        l_pic[16+3] = l_pic[16+3+12];
        l_pic[16+3+12] = l_pic[16+3+8];
        l_pic[16+3+8] = l_pic[16+3+4];
        l_pic[16+3+4] = temp;
        
        //mix columns
        if(i<9){
            for(int j=0; j < 4; j++){
                res[0] = 0x00;
                res[1] = 0x00;
                res[2] = 0x00;
                res[3] = 0x00;
                res[4] = 0x00;
                res[5] = 0x00;
                res[6] = 0x00;
                res[7] = 0x00;
                
                for(int k = 0; k < 4; k++){
                    //res[0] gf2_mul
                    temp = 0x00; 
                    temp0 = mcMatrix[0+k];
                    temp1 = l_pic[j*4+k];
                    for(;temp1;temp1>>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[0] = res[0] ^ temp;
                    
                    temp = 0x00; 
                    temp0 = mcMatrix[0+k];
                    temp1 = l_pic[16+(j*4+k)];
                    for(;temp1;temp1>>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[4+0] = res[4+0] ^ temp;
                    
                    //res[1] gf2_mul
                    temp = 0x00; 
                    temp0 = mcMatrix[4+k];
                    temp1 = l_pic[j*4+k];
                    for(;temp1;temp1 >>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[1] = res[1] ^ temp;
                    
                    temp = 0x00; 
                    temp0 = mcMatrix[4+k];
                    temp1 = l_pic[16+(j*4+k)];
                    for(;temp1;temp1 >>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[4+1] = res[4+1] ^ temp;
                    
                    //res[2] gf2_mul
                    temp = 0x00; 
                    temp0 = mcMatrix[8+k];
                    temp1 = l_pic[j*4+k];
                    for(;temp1;temp1 >>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[2] = res[2] ^ temp;
                    
                    temp = 0x00; 
                    temp0 = mcMatrix[8+k];
                    temp1 = l_pic[16+(j*4+k)];
                    for(;temp1;temp1 >>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[4+2] = res[4+2] ^ temp;
                    
                    //res[3] gf2_mul
                    temp = 0x00; 
                    temp0 = mcMatrix[12+k];
                    temp1 = l_pic[j*4+k];
                    for(;temp1;temp1 >>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[3] = res[3] ^ temp;
                    
                    temp = 0x00; 
                    temp0 = mcMatrix[12+k];
                    temp1 = l_pic[16+(j*4+k)];
                    for(;temp1;temp1 >>=1){
                        if(temp1 & 1)
                            temp ^= temp0;
                        if(temp0 & 0x80)
                            temp0 = (temp0 << 1) ^ 0x1b;
                        else
                            temp0 <<= 1;
                    }
                    res[4+3] = res[4+3] ^ temp;
                }
                l_pic[(j*4)+0] = res[0];
                l_pic[(j*4)+1] = res[1];
                l_pic[(j*4)+2] = res[2];
                l_pic[(j*4)+3] = res[3];
                
                l_pic[16+((j*4)+0)] = res[4];
                l_pic[16+((j*4)+1)] = res[5];
                l_pic[16+((j*4)+2)] = res[6];
                l_pic[16+((j*4)+3)] = res[7];
            }
            
        }
        //add round key
        for(int j = 0; j < 16; j++){
            temp = keys_s[((i+1)*16)+j];
            l_pic[j] = l_pic[j] ^ temp;
            l_pic[16+j] = l_pic[16+j] ^ temp;
        }
       //end of of current round of encryption
    }
    //end of all encryption
    //move local bytes back to global memory
    
    for(int i = 0; i < 32; i++){
        r_pic[tx+i] =  l_pic[i];
    }
    }
}



void generateRoundKeys(unsigned char*el_keys, unsigned char *el_sbox, unsigned char *rconTable){
    int currKeyIdx;
    int prevKeyIdx;
    for(int i = 0; i < 10; i++){
        
        currKeyIdx = 16*(i+1);
        prevKeyIdx = 16*i;
        
        el_keys[currKeyIdx+0] = el_keys[prevKeyIdx+13];
        el_keys[currKeyIdx+1] = el_keys[prevKeyIdx+14];
        el_keys[currKeyIdx+2] = el_keys[prevKeyIdx+15];
        el_keys[currKeyIdx+3] = el_keys[prevKeyIdx+12];
        
        el_keys[currKeyIdx+0] = el_sbox[el_keys[currKeyIdx+0]];
        el_keys[currKeyIdx+1] = el_sbox[el_keys[currKeyIdx+1]];
        el_keys[currKeyIdx+2] = el_sbox[el_keys[currKeyIdx+2]];
        el_keys[currKeyIdx+3] = el_sbox[el_keys[currKeyIdx+3]];
        
        el_keys[(16*(i+1))+0] = el_keys[(16*i)+0] ^ el_keys[(16*(i+1))+0] ^ rconTable[i];
        el_keys[(16*(i+1))+1] = el_keys[(16*i)+1] ^ el_keys[(16*(i+1))+1];
        el_keys[(16*(i+1))+2] = el_keys[(16*i)+2] ^ el_keys[(16*(i+1))+2];
        el_keys[(16*(i+1))+3] = el_keys[(16*i)+3] ^ el_keys[(16*(i+1))+3];
        
        for(int j = 1; j < 4; j++){
            el_keys[currKeyIdx+(0+(4*j))] = el_keys[currKeyIdx+(0+(4*(j-1)))] ^ el_keys[prevKeyIdx+(0+(4*j))];
            el_keys[currKeyIdx+(1+(4*j))] = el_keys[currKeyIdx+(1+(4*(j-1)))] ^ el_keys[prevKeyIdx+(1+(4*j))];
            el_keys[currKeyIdx+(2+(4*j))] = el_keys[currKeyIdx+(2+(4*(j-1)))] ^ el_keys[prevKeyIdx+(2+(4*j))];
            el_keys[currKeyIdx+(3+(4*j))] = el_keys[currKeyIdx+(3+(4*(j-1)))] ^ el_keys[prevKeyIdx+(3+(4*j))];
        }
        
    }    
}


void sequentialAES(unsigned char* input, unsigned char* output, unsigned char* keys, unsigned char* sbox, int size){
    
    unsigned char mcMatrix[] = {
        0x02, 0x03, 0x01, 0x01,
        0x01, 0x02, 0x03, 0x01,
        0x01, 0x01, 0x02, 0x03,
        0x03, 0x01, 0x01, 0x02
    };
    
    unsigned char currInput[16];
    unsigned char temp, temp0, temp1;
    unsigned char res[4];
    
    
    int currBaseIdx = 0;
    
    for(int i = 0; i < (size/16) ;i++){
    //for(int i = 0; i < 1; i++){
        currBaseIdx = i*16;
        
        //load in current input section
        for(int j = 0; j < 16; j++){
            currInput[j] = input[currBaseIdx+j];
        }
        /*printf("\ninitial input in seq function: ");
        for(int j=0; j < 16; j++){
            printf("%x ", currInput[j]);
        }*/
        
        //initial add round key
        for(int j = 0; j < 16; j++){
            currInput[j] = currInput[j] ^ keys[j];
        }
        
        /*printf("\nafter init add round key (seq): ");
        for(int j=0; j < 16; j++){
            printf("%x ", currInput[j]);
        }*/
        //begin main aes loop (10 rounds)
        for(int j = 0; j < 10; j++){
            
            //sub bytes
            for(int k = 0; k < 16; k++){
                currInput[k] = sbox[currInput[k]];
            }
            
            /*printf("\nafter sub bytes (seq): ");
            for(int k=0; k < 16; k++){
                printf("%x ", currInput[k]);
            } */
            //shift rows
            temp = currInput[1];
            currInput[1] = currInput[1+4];
            currInput[1+4] = currInput[1+8];
            currInput[1+8] = currInput[1+12];
            currInput[1+12] = temp;
            
            temp = currInput[2];
            currInput[2] = currInput[2+8];
            currInput[2+8] = temp;
            temp = currInput[2+4];
            currInput[2+4] = currInput[2+12];
            currInput[2+12] = temp;
            
            temp = currInput[3];
            currInput[3] = currInput[3+12];
            currInput[3+12] = currInput[3+8];
            currInput[3+8] = currInput[3+4];
            currInput[3+4] = temp;
            
            /*printf("\nafter shift rows (seq): ");
            for(int k=0; k < 16; k++){
                printf("%x ", currInput[k]);
            }*/
            //mix columns
            
            if(j<9){
                
                for(int k = 0; k < 4; k++){
                    
                    res[0] = 0x00;
                    res[1] = 0x00;
                    res[2] = 0x00;
                    res[3] = 0x00;
                    
                    for(int l = 0; l < 4; l++){
                    
                        temp = 0x00; 
                        temp0 = mcMatrix[0+l];
                        temp1 = currInput[k*4+l];
                        
                        for(;temp1;temp1>>=1){
                            if(temp1 & 1)
                                temp ^= temp0;
                            if(temp0 & 0x80)
                                temp0 = (temp0 << 1) ^ 0x1b;
                            else
                                temp0 <<= 1;
                        }
                        res[0] = res[0] ^ temp;
                    
                    //res[1] gf2_mul
                        temp = 0x00; 
                        temp0 = mcMatrix[4+l];
                        temp1 = currInput[k*4+l];
                        
                        for(;temp1;temp1 >>=1){
                            if(temp1 & 1)
                                temp ^= temp0;
                            if(temp0 & 0x80)
                                temp0 = (temp0 << 1) ^ 0x1b;
                            else
                                temp0 <<= 1;
                        }
                        res[1] = res[1] ^ temp;
                    
                    //res[2] gf2_mul
                        temp = 0x00; 
                        temp0 = mcMatrix[8+l];
                        temp1 = currInput[k*4+l];
                        
                        for(;temp1;temp1 >>=1){
                            if(temp1 & 1)
                                temp ^= temp0;
                            if(temp0 & 0x80)
                                temp0 = (temp0 << 1) ^ 0x1b;
                            else
                                temp0 <<= 1;
                        }
                        res[2] = res[2] ^ temp;
                    
                    //res[3] gf2_mul
                        temp = 0x00; 
                        temp0 = mcMatrix[12+l];
                        temp1 = currInput[k*4+l];
                        
                        for(;temp1;temp1 >>=1){
                            if(temp1 & 1)
                                temp ^= temp0;
                            if(temp0 & 0x80)
                                temp0 = (temp0 << 1) ^ 0x1b;
                            else
                                temp0 <<= 1;
                        }
                        res[3] = res[3] ^ temp;
                    }
                    
                    currInput[(k*4)+0] = res[0];
                    currInput[(k*4)+1] = res[1];
                    currInput[(k*4)+2] = res[2];
                    currInput[(k*4)+3] = res[3];
                }
            }
            
            /*printf("\nafter mis cols (seq): ");
            for(int k=0; k < 16; k++){
                printf("%x ", currInput[k]);
            }*/
            
            //add round key
            for(int k = 0; k < 16; k++){
                currInput[k] = currInput[k] ^ keys[(j+1)*16+k];
            }
            
            /*printf("\nafter add round key (seq): ");
            for(int k=0; k < 16; k++){
                printf("%x ", currInput[k]);
            }*/
            
        }
        //move final encrypted 16 byte block t correcto output index
        for(int j = 0; j < 16; j++){
            output[currBaseIdx+j] = currInput[j];
        }
    }
}

void generateRoundKey(unsigned char* prevKey, unsigned char* newKey, int round, unsigned char* sbox,
    unsigned char* rconTable){
    
    //rotWord    
    newKey[0] = prevKey[13];
    newKey[1] = prevKey[14];
    newKey[2] = prevKey[15];
    newKey[3] = prevKey[12];
    
    //subword
    newKey[0] = sbox[newKey[0]];
    newKey[1] = sbox[newKey[1]];
    newKey[2] = sbox[newKey[2]];
    newKey[3] = sbox[newKey[3]];
    
    //rcon
    newKey[0] = prevKey[0] ^ newKey[0] ^ rconTable[round] ;
    newKey[1] = prevKey[1] ^ newKey[1];
    newKey[2] = prevKey[2] ^ newKey[2];
    newKey[3] = prevKey[3] ^ newKey[3];
    
    for(int i = 1; i< 4; i++){
        newKey[0+(4*i)] = newKey[0+(4*(i-1))] ^ prevKey[0+(4*i)];
        newKey[1+(4*i)] = newKey[1+(4*(i-1))] ^ prevKey[1+(4*i)];
        newKey[2+(4*i)] = newKey[2+(4*(i-1))] ^ prevKey[2+(4*i)];
        newKey[3+(4*i)] = newKey[3+(4*(i-1))] ^ prevKey[3+(4*i)];
    }
    
}

int main (int argc, char **argv){
    unsigned char sbox[] ={
        0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
        0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
        0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
        0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
        0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
        0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
        0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
        0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
        0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
        0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
        0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
        0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
        0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
        0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
        0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
        0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
    };
    
    unsigned char rconTable[]={0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36};
    
    
    int thresh = 8000;
    char *filename;
    filename = strdup(DEFAULT_FILENAME);
    
    if(argc > 1){
        if(argc == 2){
            filename = strdup(argv[1]);
        } 
        fprintf(stderr, "file %s    threshold %d\n", filename, thresh);
    }
    
    int xsize, ysize, maxval;
    unsigned char *pic = read_ppm(filename, &xsize, &ysize, &maxval);
    
    printf("xsize: %d, ysize: %d, maxval: %d\n", xsize, ysize, maxval);
    //zero out the pic first because you need to know how much of the array is actually being used
    
    int numbytes = xsize * ysize * 3 * sizeof(int);
    //int numbytesp = xsize *ysize * 3 *sizeof(int)*3;
    
    //int *result = (int *) malloc(numbytes);
    
    //unsigned char *resultc = (unsigned char *) malloc(numbytes*3);
    
    printf("size of int: %ld", sizeof(int));
    unsigned char *resultc = (unsigned char *) malloc(xsize*ysize*3);
    if(!resultc){
        fprintf(stderr, "sobel() unable to malloc %d bytes\n", numbytes); 
        exit(-1);
    }
    
    //memset(resultc, 0x00, numbytes*3);
    memset(resultc, 0x00, xsize*ysize*3);
    
    //memset(resultc, 0, xsize * ysize*sizeof(int)*3);
    
    printf("got here yurd baby 3\n");
    //int i, j, offset;
    //char temp[3];
    //please kill me
    printf("got here 1 14/16\n");
    printf("sizes: %d, %d\n",xsize,ysize);
    
    
    /*for(i = 0; i< ysize; i++){
        for(j = 0; j < xsize; j++){
            offset = i * xsize + j;
            //offset = offset +1 -1;
            
            //printf("%x,%x,%x ", pic[offset*3],pic[offset*3+1],pic[offset*3+2]);
            resultc[offset*3] = pic[offset*3];
            //resultc[offset*3] = 0x00;
            resultc[offset*3+1] = pic[offset*3+1];
            //resultc[offset*3+1] = 0x00;
            //resultc[offset*3+2] = pic[offset*3+2];
            resultc[offset*3+2] = pic[offset*3+2];
            resultc[offset*3+2] = 0x80;
            //result = (int*)(pic[offset*3])
            //((*int)&temp[0]) = (pic[offset]);
            //printf("%x,%x ",temp[0],temp[1]);
            //itoa(pic[offset], temp, 16);
            //printf("%s, ",temp);
            
            //temp[0]= *((char*)pic[offset]);
            //temp[1]= *(((char*)pic[offset+1]));
            //temp[2]= *((char*)pic[offset]);
            //printf("(%x,%x,%x) ",temp[0], temp[1], temp[2]);
            
            
                
            //magnitude = sum1*sum1 + sum2*sum2;
            
            //if(magnitude > thresh)
                //result[offset] = 128;
            //else
                //result[offset] = 0;
        }
        //printf("\n");
    }*/
    /*printf("\nxsize * ysize * 3 * sizeof(int)*3 - 1: %x\n xsize * ysize * 3 * sizeof(int)-1: %x\n xsize * ysize*3 -1:%x\n(xsize * ysize) -1: %x\n",
        resultc[(xsize*ysize*3*sizeof(int)*3)-5], resultc[(xsize*ysize*3*sizeof(int))-5], 
        resultc[(xsize*ysize*3)-5], resultc[(xsize*ysize)-5]);*/
        
       //////////   BEGINNING OF SEQUENTIAL AES    ////////////////////////////////////////////////////////////////// 
        
        
        printf("beginning (fixed?): ");
        for(int i = 0; i < 16; i++){
            printf("%x ", pic[i]);
        }
        printf("\n");
    //GENERATE KEY!
    
    //unsigned char roundKeys[11][16];
    unsigned char roundKeys2 [176]; //11*16 = 176 quick maths
    //unsigned char roundKeyHolder...
    
    /*unsigned char testKey[] = {
        0x21, 0x8f, 0x0d, 0x3e, 
        0x95, 0x3c, 0xb0, 0x19,
        0xd1, 0x63, 0x4b, 0xf4,
        0x12, 0xab, 0x8f, 0x3c
    };*/
    
    unsigned char testKey2[] = {
        0x2b, 0x7e, 0x15, 0x16,
        0x28, 0xae, 0xd2, 0xa6,
        0xab, 0xf7, 0x15, 0x88,
        0x09, 0xcf, 0x4f, 0x3c
    };
   /* 
    for(int i = 0; i< 16; i++){
        roundKeys[0][i] = testKey2[i];
    }*/
    for(int i = 0; i < 16; i++){
        roundKeys2[i] = testKey2[i];
    }
    
    generateRoundKeys(roundKeys2, sbox, rconTable);
    
    for(int i = 0; i < 11; i++){
        printf("k%d: ",i);
        for(int j = 0; j < 16; j++){
            printf("%x ",roundKeys2[(i*16)+j]);
        }
        printf("\n");
    }
    
    //clock_t start_s, end_s;
    //double cpu_time_used;
    
    //start_s=clock();
    hipEvent_t start_s, stop_s;
    float deltaTime_s;
    
    hipEventCreate(&start_s);
    hipEventCreate(&stop_s);
    hipEventRecord(start_s, 0);
    
    sequentialAES(pic, resultc, roundKeys2, sbox, xsize*ysize*3);
    
    hipEventRecord(stop_s,0);
    hipEventSynchronize(stop_s);
    hipEventElapsedTime(&deltaTime_s,start_s,stop_s);
    hipEventDestroy(start_s);
    hipEventDestroy(start_s);
    
    //end_s= clock();
    //cpu_time_used = ((double) (end_s-start_s)) /CLOCKS_PER_SEC;
    
    //printf("Sequential AES time = %f\n", cpu_time_used);
    printf("Delta Time (seq) = %f\n", deltaTime_s);
    
    printf("sequential first 16 bytes result: ");
    for(int i =0; i < 16; i++){
        printf("%x ", resultc[i]);
    }
    printf("\n");
    //first round of generate new key
    //generateRoundKey(testKey2, roundKeys[0], 0, sbox, rconTable);
    /*for(int i = 1; i < 11; i++){
        generateRoundKey(roundKeys[i-1], roundKeys[i], i-1,sbox,rconTable);
    }*/
    //printing roundkeys
    /*for(int i = 0; i < 11; i++){
        for(int j = 0; j < 16; j++){
            printf("%x ", roundKeys[i][j]);
        }
        printf("\n");
    }*/
    
    
    unsigned char* h_output = (unsigned char*)malloc(xsize*ysize*3);
    
    unsigned char* test_host_input = (unsigned char*)malloc(16*sizeof(unsigned char));
    unsigned char* test_host_output = (unsigned char*)malloc(16*sizeof(unsigned char));
    unsigned char* h_sbox = (unsigned char*)malloc(16*16*sizeof(unsigned char));
    unsigned char* h_roundKeys = (unsigned char*)malloc(176*sizeof(unsigned char));
    //unsigned char* h_rconTable = (unsigned char*)malloc(10*sizeof(unsigned char));
    
    unsigned char* d_input;
    unsigned char* d_output;
    
    unsigned char* test_device_input;
    unsigned char* test_device_output;
    unsigned char* d_sbox;
    unsigned char* d_roundKeys;
    //unsigned char* d_rconTable;
    
    hipMalloc(&d_input, xsize*ysize*3);
    hipMalloc(&d_output, xsize*ysize*3);
    
    hipMalloc(&test_device_input, 16*sizeof(unsigned char));
    hipMalloc(&test_device_output, 16*sizeof(unsigned char));
    hipMalloc(&d_sbox, 16*16*sizeof(unsigned char));
    hipMalloc(&d_roundKeys, 176*sizeof(unsigned char));
    //cudaMalloc(&d_rconTable, 10*sizeof(unsigned char));
    
    //initialize test input
    
    test_host_input[0] = 0x56;
    test_host_input[1] = 0xf6;
    test_host_input[2] = 0x4b;
    test_host_input[3] = 0xb5;
    test_host_input[4] = 0x9c;
    test_host_input[5] = 0x7d;
    test_host_input[6] = 0x17;
    test_host_input[7] = 0xa4;
    test_host_input[8] = 0x9b;
    test_host_input[9] = 0x08;
    test_host_input[10] = 0x90;
    test_host_input[11] = 0x02;
    test_host_input[12] = 0x50;
    test_host_input[13] = 0xeb;
    test_host_input[14] = 0xbb;
    test_host_input[15] = 0x3f;
    
    for(int i = 0; i < 16*16; i++){
        h_sbox[i] = sbox[i];
    }
    for(int i = 0; i < 176; i++){
        h_roundKeys[i] = roundKeys2[i];
    }
    /*for(int i = 0; i < 10; i++){
        h_rconTable[i] = rconTable[i];
    }*/
    
    hipMemcpy(d_input, pic, xsize*ysize*3, hipMemcpyHostToDevice);
    
    
    hipMemcpy(test_device_input, test_host_input,16*sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_sbox, h_sbox,16*16*sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_roundKeys, h_roundKeys, 176*sizeof(unsigned char),hipMemcpyHostToDevice);
    //cudaMemcpy(d_rconTable, h_rconTable, cudaMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    float deltaTime;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    aes_moreWork<<<(int)floor((float)(xsize*ysize*3)/(BLOCK_SIZE*16)),BLOCK_SIZE>>>(d_input, d_output, d_roundKeys, d_sbox, xsize*ysize*3);
    
    //aes_moreWork<<<(int)floor((float)(xsize*ysize*3)/(BLOCK_SIZE*32)),BLOCK_SIZE>>>(d_input, d_output, d_roundKeys, d_sbox, xsize*ysize*3);
    printf("\n");
    //aes_moreWork<<<1,1>>>(d_input, d_output, d_roundKeys, d_sbox);
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&deltaTime,start,stop);
    hipEventDestroy(start);
    hipEventDestroy(start);
    //aes<<<1,1>>>(test_device_input,test_device_output, d_roundKeys, d_sbox);
    
    hipMemcpy(h_output,d_output, xsize*ysize*3, hipMemcpyDeviceToHost);
    //cudaMemcpy(test_host_output, test_device_output, 16*sizeof(unsigned char), cudaMemcpyDeviceToHost);
    
    printf("Delta Time (more work) = %f\n", deltaTime);
    
    printf("\nFinal Output!: ");
    for(int i = 0; i < 16; i++){
        printf("%x ",h_output[i]);
    }
    
    printf("got here 2 son\n");
    write_ppm((unsigned char *) "result8sendhelp.ppm", xsize, ysize, 255, resultc);
    write_ppm((unsigned char *) "encrypted_gary.ppm", xsize,ysize, 255, h_output);
    
    int mismatchCounter = 0;
    for(int i =0 ; i < xsize*ysize*3; i++){
        if(resultc[i] != h_output[i]){
            mismatchCounter++;
        }
    }
    printf("mismatchCounter: %d\n", mismatchCounter);
    
    free(pic);
    free(resultc);
    free(test_host_input);
    free(test_host_output);
    free(h_sbox);
    free(h_output);
    //free(h_)
    
    hipFree(test_device_input);
    hipFree(test_device_output);
    hipFree(d_sbox);
    hipFree(d_output);
    hipFree(d_input);
    
    fprintf(stderr, "aes done\n");
}


